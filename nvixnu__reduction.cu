#include "hip/hip_runtime.h"
#include "nvixnu__reduction.h"

__global__
void nvixnu__sum_by_block(float *v, float *sum){
    extern __shared__ float partial_sum[];
    unsigned int tx = threadIdx.x;
    // Copies the elements to be added to the shared memory
    partial_sum[tx] = v[blockIdx.x * blockDim.x + tx];

    // Halve the stride in each iteration, bringing the temporary sums into the first half
    for(unsigned int stride = blockDim.x/2; stride >= 1; stride /= 2){
        __syncthreads();
        if(tx < stride){ // Check if thread is inthe first half
            partial_sum[tx] += partial_sum[tx+stride];
        }
    }
    __syncthreads();
    sum[blockIdx.x] = partial_sum[0];
}