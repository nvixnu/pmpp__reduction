#include "hip/hip_runtime.h"
#include "nvixnu__reduction.h"

__global__
void sum_by_block(double *v, double *sum, const int length){
    extern __shared__ double partial_sum[];
    unsigned int tx = threadIdx.x;
    int tid = blockIdx.x * blockDim.x + tx;

    // Copies the elements to be added to the shared memory. If the value is a garbage one, includes zero instead
    partial_sum[tx] = tid < length ? v[tid] : 0.0;

    // Halve the stride in each iteration, bringing the temporary sums into the first half
    for(unsigned int stride = blockDim.x/2; stride >= 1; stride /= 2){
        __syncthreads();
        if(tx < stride){ // Check if thread is inthe first half
            partial_sum[tx] += partial_sum[tx+stride];
        }
    }
    __syncthreads();
    sum[blockIdx.x] = partial_sum[0];    
}