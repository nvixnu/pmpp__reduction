#include "hip/hip_runtime.h"
#include "nvixnu__reduction.h"

__global__
void nvixnu__sum_by_block(float *v, float *sum){
    extern __shared__ float partial_sum[];
    unsigned int tx = threadIdx.x;

    partial_sum[threadIdx.x] = v[blockIdx.x * blockDim.x + tx];

    for(unsigned int stride = blockDim.x/2; stride >= 1; stride /= 2){
        __syncthreads();
        if(tx < stride){
            partial_sum[tx] += partial_sum[tx+stride];
        }
    }
    __syncthreads();
    sum[blockIdx.x] = partial_sum[0];
}